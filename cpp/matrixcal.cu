#include "hip/hip_runtime.h"
#include "matrixcal.h"


//TODO: change to device_vector
/*__device__
void find_neighbour(int i, int m, int n, thrust::device_vector<int> res){
    res.clear();
    int idxes[4] = {i-m,i+m,i-1,i+1};
    bool exist[4] = {true,true,true,true};
    if(i % m == 0){
        exist[2] = false;
    }
    if((i % n) == (n - 1)){
        exist[3] = false;
    }
    if(i < m){
        exist[0] = false;
    }
    if(i > (n*m-1-m)){
        exist[1] = false;
    }
    for(int s = 0; s < 4; s ++){
        if(exist[s]){
            res.push_back(idxes[s]);
        }
    }
}*/



//TODO:gpu
__global__
void mul_kernel(double *a, double *b, double *c, int m, int n, int p){
    int index = threadIdx.x;
    int stride = blockDim.x;
    double tmp;
    for(int x = index; x < m; x += stride){
        for(int y = 0; y < p; y ++){
            tmp = 0;
            for(int t = 0; t < n; t++){
                tmp += (a[x+t*m] * b[t+y*n]);
            }
            c[x+y*m] = tmp;
        }
    }
}


//TODO:change to cuda
__global__
void adjacency_mul_kernel(double *x, double *res, int rowx, int colx, int srcx, int srcy){
    int index = threadIdx.x;
    int stride = blockDim.x;
    for(int i=index; i < rowx; i += stride){
        //find_neighbour(i, srcx,srcy, neighs);

    thrust::device_vector<int> neighs;
    int m = srcx;
    int n = srcy;
    int idxes[4] = {i-m,i+m,i-1,i+1};
    bool exist[4] = {true,true,true,true};
    if(i % m == 0){
        exist[2] = false;
    }
    if((i % n) == (n - 1)){
        exist[3] = false;
    }
    if(i < m){
        exist[0] = false;
    }
    if(i > (n*m-1-m)){
        exist[1] = false;
    }
    for(int s = 0; s < 4; s ++){
        if(exist[s]){
            neighs.push_back(idxes[s]);
        }
    }

        int neigh_size = neighs.size();
        for (int j = 0; j < colx; j++){
            double val = 0.0f;
            for (int vstart = 0; vstart < neigh_size; vstart++){
                val += (x[j*rowx+neighs[vstart]]);
            }
            val -= (neigh_size * x[j*rowx + i]);
            //std::cout << j*rowx+i << std::endl;
            res[j*rowx+i] = val;
        }
    }
}


//TODO: change to gpu
__global__
void matrix_sub_kernel(double* a,double* b,double* c,double* res, int size){
    int index = threadIdx.x;
    int stride = blockDim.x;
    for(int i=index; i < size; i += stride){
        res[i] = a[i] - b[i] - c[i];
    }
}


//TODO: change to gpu
double dot_kernel(double *a,double *b, int size){
    double res = 0.0f;
    for(int i = 0; i < size; i ++){
        res += (a[i]*b[i]);
    }
    return res;
}



//TODO: change to gpu
__global__
void matrix_add_kernel(double* a,double* b,double* res, int size){
    int index = threadIdx.x;
    int stride = blockDim.x;
    for(int i=index; i < size; i += stride){
        res[i] = a[i] + b[i] ;
    }
}


//TODO: change to gpu
__global__
void matrix_add_scale_kernel(double* a,double* b,double scale,double* res, int size){
    int index = threadIdx.x;
    int stride = blockDim.x;
    for(int i=index; i < size; i += stride){
        res[i] = a[i] + scale*b[i] ;
    }
}



void randomMatrix(Matrix &m){
    for (int i = 0; i < m.getrow(); i ++){
        for (int j = 0; j < m.getcol(); j ++){
            m.set(i,j,rand() % 10);
        }
    }
}

int load_from_file(std::string fname, Matrix &m){
    std::ifstream infile(fname);
    int i = 0;
    double t = 0.0000000000000000000;
    while(infile >> t){
        m.set_by_idx(i,t);
        i++;
    }
    return 0;
}

int load_diagonal(std::string fname, Matrix &m){
    std::ifstream infile(fname);
    int i = 0;
    double t = 0.0000000000000000000;
    while(infile >> t){
        m.set_diagonal(i,t);
        i++;
    }
    return 0;

}

void mul(Matrix &a, Matrix &b, Matrix &res){
    int rowa, rowb,cola, colb;
    rowa = a.getrow();
    rowb = b.getrow();
    cola = a.getcol();
    colb = b.getcol();

    if (cola != rowb){
        throw "matrix do not match in matrix multiplication";
    }

    res.alloc(rowa,colb);
    int blockSize = 1024;
    int blocks = (rowa + blockSize - 1) / blockSize;
    mul_kernel<<<blocks,blockSize>>>(a.get_data(),b.get_data(), res.get_data(), rowa,cola,colb);
}



/*
 * kron_mul perform reshape and kronecker product, the caller should perform transpose of the left matrix by itself.
 */
void kron_mul(Matrix &ma, Matrix &mb, Matrix &mx, Matrix &res){
    if(mx.getcol() != 1){
        throw "the middle matrix should have one column when performing kronecker reshape product";
    }
    int bcol, xrow, arow; 
    arow = ma.getrow();
    xrow = mx.getrow();
    bcol = mb.getcol();

    if (arow * bcol != xrow){
        throw "the matrix sizes do not match when performing kronecker reshape product";
    }
    //res.alloc(leftx, righty);
    
    if (mx.resize(bcol,arow) != 0){
        throw "matrix resize failed";
    }
    Matrix tmpres;
    if (mb.is_identity()){
        tmpres.copy(mx);
    }else{
        mul(mb,mx,tmpres);
    }

    if (ma.is_identity()){
        res.copy(tmpres);
    }else{
        mul(tmpres, ma, res);
    }

    res.resize(xrow,1);
    return;
}

void kron_prod(Matrix &a, Matrix &b, Matrix &res){
    int arow,acol, brow,bcol;
    arow = a.getrow();
    acol = a.getcol();
    brow = b.getrow();
    bcol = b.getcol();

    res.alloc(arow*brow, acol*bcol);

    for(int outi = 0; outi < arow; outi ++){
        for(int outj = 0; outj < acol; outj ++){
            for(int inneri = 0; inneri < brow; inneri ++){
                for(int innerj = 0; innerj < bcol; innerj ++){
                    res.set(outi * brow+inneri, outj*bcol + innerj, a.get(outi,outj) * b.get(inneri,innerj));
                }
            }
        }
    }
}


void adjacency_mul(Matrix& x, Matrix& res, int rowx, int colx, int srcx,int srcy){

    res.alloc(rowx*colx,1);
    //std::cout << "full size" << rowx*colx << std::endl;
    double* rawx = x.get_data();
    double* rawres = res.get_data();
    //res.alloc(rowx,colx);
    int blockSize = 1024;
    int blocks = (rowx + blockSize - 1) / blockSize;
    adjacency_mul_kernel<<<blocks,blockSize>>>(rawx,rawres,rowx,colx,srcx,srcy);
}



void matrix_sub(Matrix& a,Matrix& b,Matrix& c,Matrix& res){
    int arow = a.getrow();
    int acol = a.getcol();
    int brow = b.getrow();
    int bcol = b.getcol();
    int crow = c.getrow();
    int ccol = c.getcol();
    if (arow != brow || arow != crow){
        throw "matrix sub failed.";
    }
    if (acol != bcol || acol != ccol){
        throw "matrix sub failed.";
    }
    res.alloc(arow,acol);
    int blockSize = 1024;
    int blocks = (arow*acol + blockSize - 1) / blockSize;
    matrix_sub_kernel<<<blocks,blockSize>>>(a.get_data(),b.get_data(),c.get_data(),res.get_data(),arow*acol);
}


double dot(Matrix &a,Matrix &b){
    if (a.getcol() != 1||b.getcol() != 1 || a.getrow() != b.getrow()) {
        throw "dot product failed.";
    }
    return dot_kernel(a.get_data(),b.get_data(),a.getrow());
}



void matrix_add(Matrix& a,Matrix& b,Matrix& res){
    int arow = a.getrow();
    int acol = a.getcol();
    int brow = b.getrow();
    int bcol = b.getcol();
    if (arow != brow){
        throw "matrix sub failed.";
    }
    if (acol != bcol){
        throw "matrix sub failed.";
    }
    res.alloc(arow,acol);
    int blockSize = 1024;
    int blocks = (arow*acol + blockSize - 1) / blockSize;
    matrix_add_kernel<<<blocks,blockSize>>>(a.get_data(),b.get_data(),res.get_data(),arow*acol);
}


void matrix_add_scale(Matrix& a,Matrix& b,double scale,Matrix &res){
    int arow = a.getrow();
    int acol = a.getcol();
    int brow = b.getrow();
    int bcol = b.getcol();
    if (arow != brow){
        throw "matrix sub failed.";
    }
    if (acol != bcol){
        throw "matrix sub failed.";
    }
    int blockSize = 1024;
    int blocks = (arow*acol + blockSize - 1) / blockSize;
    matrix_add_scale_kernel<<<blocks,blockSize>>>(a.get_data(),b.get_data(),scale,res.get_data(),arow*acol);
}


