#include "matrix.h"

Matrix::Matrix(int x, int y){
    row = x;
    col = y;
    hipMallocManaged(&data, x*y*sizeof(float));
}

Matrix::~Matrix(){
    hipFree(data);
}

float* Matrix::getdata(){
    return data;
}

void Matrix::print(){
    for (int i=0; i < row; i++) {
	std::cout << "[";
        for (int j=0; j < col; j ++){
		std::cout << data[i*col+j] << " ";
        }
	std::cout << "]" << std::endl; 
    }
}


