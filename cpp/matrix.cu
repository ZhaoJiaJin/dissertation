#include "matrix.h"
#include <iostream.h>

Matrix::Matrix(int x, int y){
    row = x;
    col = y;
    hipMallocManaged(&data, x*y*sizeof(float));
}

Matrix::~dev_array(){
    cubaFree(x);
}

float* Matrix::getdata(){
    return x
}

void Matrix::print(){
    for (int i=0; i < row; i++) {
        std::cout << "[";
        for (int j=0; j < col; j ++){
            std::cout << x[i*col+j] << " ";
        }
        std::cout << "]" << endl;
    }
}


